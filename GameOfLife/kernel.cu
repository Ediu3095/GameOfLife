#include "hip/hip_runtime.h"
﻿#include <iostream>

#include "kernel.cuh"

__global__ void updateKernel(const int rows, const int cols, const bool* in, bool* out)
{
	// Get the row and column with which to work
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	// Stop threads that overstep the bounds of the grid
	if (i >= rows || j >= cols)
		return;

	// Count the amount of living neighbours around this position
	int neighbours = -in[i * cols + j];
	for (int ioff = -1; ioff <= 1; ioff++)
		for (int joff = -1; joff <= 1; joff++)
		{
			// Handle out of bounds indices
			int iaux = i + ioff, jaux = j + joff;

			if (iaux < 0)
				iaux = rows - 1;
			else if (iaux >= rows)
				iaux = 0;

			if (jaux < 0)
				jaux = cols - 1;
			else if (jaux >= cols)
				jaux = 0;

			// Count a new neighbour
			neighbours += in[iaux * cols + jaux];
		}

	// Store the amount of neighbours in the output
	out[i * cols + j] = (neighbours == 3 || (neighbours == 2 && in[i * cols + j]));
}

hipError_t updateWithCuda(const int rows, const int cols, const bool* in, bool* out)
{
	bool* dev_in;
	bool* dev_out;
	hipError_t cudaStatus;

	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		}

		// Allocate GPU buffers for three vectors (two input, one output).
		cudaStatus = hipMalloc((void**)&dev_in, rows * cols * sizeof(bool));
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&dev_out, rows * cols * sizeof(bool));
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("hipMalloc failed!");
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_in, in, rows * cols * sizeof(bool), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("hipMemcpy failed!");
		}

		// Launch a kernel on the GPU with one thread for each element.
		int temp = std::min(rows, cols);
		float tileSize = (temp < 32) ? temp : 32;

		dim3 gridSize(std::ceil(rows / tileSize), std::ceil(cols / tileSize));
		dim3 blockSize(tileSize, tileSize);
		updateKernel <<<gridSize, blockSize>>>(rows, cols, dev_in, dev_out);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			//fprintf(stderr, "updateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw std::exception("updateKernel launch failed");
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			//fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			throw std::exception("hipDeviceSynchronize returned an error code after launching addKernel!");
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(out, dev_out, rows * cols * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			throw std::exception("hipMemcpy failed!");
		}
	}
	catch (std::exception ex)
	{
		fprintf(stderr, ex.what());
	}

	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}
